#include "hip/hip_runtime.h"
//
// Header PROB_FUNCTION_INFO_FRS_FLZONO_H Begin
//

#ifndef PROB_FUNCTION_INFO_FRS_FLZONO_H
#define PROB_FUNCTION_INFO_FRS_FLZONO_H

#include "hip/hip_runtime.h"
#include ""

#define MAX_ZONO_NUM 1200

__constant__ int dev_num_zono[1];
__constant__ int dev_grid_size[1];
__constant__ double dev_dx[MAX_ZONO_NUM];
__constant__ double dev_dy[MAX_ZONO_NUM];
__constant__ double dev_rot_angle[MAX_ZONO_NUM];
__constant__ double dev_g_p_x[MAX_ZONO_NUM];
__constant__ double dev_p_slice_beta[1];
__constant__ double dev_cg_p[2];

// probablity density function
__device__ inline void f_fun(double& f, double* x, double mu1, double mu2,
                             double s1_1, double s2_1, double s2_2,
                             int zono_idx) {
  double x1 = x[0];
  double x2 = x[1];

  //     double mu1 = mu_sigma[0];
  //     double mu2 = mu_sigma[1];
  //     double s1_1 = mu_sigma[2];
  //     double s2_1 = mu_sigma[3];
  //     double s2_2 = mu_sigma[4];

  double R1_1 = cos(dev_rot_angle[zono_idx]);
  double R1_2 = -sin(dev_rot_angle[zono_idx]);
  double R2_1 = -R1_2;
  double R2_2 = R1_1;

  double t2 = s1_1 * s2_2;
  double t3 = s2_1 * s2_1;
  double t4 = mu1 / 2.0;
  double t5 = mu2 / 2.0;
  double t6 = (R1_1 * x1) / 2.0;
  double t7 = (R1_2 * x1) / 2.0;
  double t8 = (R2_1 * x2) / 2.0;
  double t9 = (R2_2 * x2) / 2.0;
  double t10 = -t3;
  double t11 = -t4;
  double t12 = -t5;
  double t13 = t2 + t10;
  double t15 = t6 + t8 + t11;
  double t16 = t7 + t9 + t12;
  double t14 = 1.0 / t13;
  f = (1.0 / sqrt(t13) *
       exp(-(s1_1 * t14 * t16 - s2_1 * t14 * t15) *
               (-mu2 + R1_2 * x1 + R2_2 * x2) +
           (s2_1 * t14 * t16 - s2_2 * t14 * t15) *
               (-mu1 + R1_1 * x1 + R2_1 * x2))) /
      (3.141592653589793 * 2.0);
}

// first order derivative of probablity density function
__device__ inline void Jacobian_fun(double* J, double* x, double mu1,
                                    double mu2, double s1_1, double s2_1,
                                    double s2_2, int zono_idx) {
  double x1 = x[0];
  double x2 = x[1];

  //     double mu1 = mu_sigma[0];
  //     double mu2 = mu_sigma[1];
  //     double s1_1 = mu_sigma[2];
  //     double s2_1 = mu_sigma[3];
  //     double s2_2 = mu_sigma[4];

  double R1_1 = cos(dev_rot_angle[zono_idx]);
  double R1_2 = -sin(dev_rot_angle[zono_idx]);
  double R2_1 = -R1_2;
  double R2_2 = R1_1;

  double t2 = R1_1 * x1;
  double t3 = R1_2 * x1;
  double t4 = R2_1 * x2;
  double t5 = R2_2 * x2;
  double t6 = s1_1 * s2_2;
  double t7 = s2_1 * s2_1;
  double t8 = 1.0 / 3.141592653589793;
  double t9 = -mu1;
  double t10 = -mu2;
  double t11 = mu1 / 2.0;
  double t12 = mu2 / 2.0;
  double t13 = t2 / 2.0;
  double t14 = t3 / 2.0;
  double t15 = t4 / 2.0;
  double t16 = t5 / 2.0;
  double t17 = -t7;
  double t18 = -t11;
  double t19 = -t12;
  double t21 = t2 + t4 + t9;
  double t22 = t3 + t5 + t10;
  double t20 = t6 + t17;
  double t25 = t13 + t15 + t18;
  double t26 = t14 + t16 + t19;
  double t23 = 1.0 / t20;
  double t24 = 1.0 / sqrt(t20);
  double t27 = s1_1 * t23 * t26;
  double t28 = s2_1 * t23 * t25;
  double t29 = s2_2 * t23 * t25;
  double t30 = s2_1 * t23 * t26;
  double t31 = -t28;
  double t32 = -t30;
  double t33 = t27 + t31;
  double t34 = t29 + t32;
  double t35 = t22 * t33;
  double t36 = t21 * t34;
  double t37 = -t35;
  double t38 = -t36;
  double t39 = t37 + t38;
  double t40 = exp(t39);
  J[0] = t8 * t24 * t40 *
         (R1_1 * t34 + R1_2 * t33 +
          t22 * ((R1_2 * s1_1 * t23) / 2.0 - (R1_1 * s2_1 * t23) / 2.0) +
          t21 * ((R1_1 * s2_2 * t23) / 2.0 - (R1_2 * s2_1 * t23) / 2.0)) *
         (-1.0 / 2.0);
  J[1] = t8 * t24 * t40 *
         (R2_1 * t34 + R2_2 * t33 +
          t22 * ((R2_2 * s1_1 * t23) / 2.0 - (R2_1 * s2_1 * t23) / 2.0) +
          t21 * ((R2_1 * s2_2 * t23) / 2.0 - (R2_2 * s2_1 * t23) / 2.0)) *
         (-1.0 / 2.0);
}

// second order derivative of probablity density function
__device__ inline void Hessian_fun(double* H, double* x, double mu1, double mu2,
                                   double s1, double s2, double s3,
                                   int zono_idx) {
  // this function only returns ddf/dxdx and ddf/dxdy.
  double x1 = x[0];
  double x2 = x[1];

  //     double mu1 = mu_sigma[0];
  //     double mu2 = mu_sigma[1];
  //     double s1 = mu_sigma[2];
  //     double s2 = mu_sigma[3];
  //     double s3 = mu_sigma[4];

  double r1 = cos(dev_rot_angle[zono_idx]);
  double r2 = sin(dev_rot_angle[zono_idx]);
  double r3 = -r2;
  double r4 = r1;

  double t2 = s1 * s3;
  double t3 = r1 * x1;
  double t4 = r2 * x2;
  double t5 = r3 * x1;
  double t6 = r4 * x2;
  double t7 = s2 * s2;
  double t8 = 1.0 / 3.141592653589793;
  double t9 = -mu1;
  double t10 = -mu2;
  double t11 = mu1 / 2.0;
  double t12 = mu2 / 2.0;
  double t13 = -t7;
  double t14 = t3 / 2.0;
  double t15 = t4 / 2.0;
  double t16 = t5 / 2.0;
  double t17 = t6 / 2.0;
  double t18 = -t11;
  double t19 = -t12;
  double t21 = t3 + t4 + t9;
  double t22 = t5 + t6 + t10;
  double t20 = t2 + t13;
  double t35 = t14 + t15 + t18;
  double t36 = t16 + t17 + t19;
  double t23 = 1.0 / t20;
  double t24 = sqrt(t20);
  double t25 = t24;
  double t27 = (r1 * s2 * t23) / 2.0;
  double t28 = (r1 * s3 * t23) / 2.0;
  double t29 = (r2 * s2 * t23) / 2.0;
  double t30 = (r3 * s1 * t23) / 2.0;
  double t31 = (r2 * s3 * t23) / 2.0;
  double t32 = (r3 * s2 * t23) / 2.0;
  double t33 = (r4 * s1 * t23) / 2.0;
  double t34 = (r4 * s2 * t23) / 2.0;
  double t41 = s2 * t23 * t35;
  double t42 = s3 * t23 * t35;
  double t43 = s1 * t23 * t36;
  double t44 = s2 * t23 * t36;
  double t26 = 1.0 / t25;
  double t37 = -t30;
  double t38 = -t32;
  double t39 = -t33;
  double t40 = -t34;
  double t45 = -t43;
  double t46 = -t44;
  double t47 = t27 + t37;
  double t48 = t28 + t38;
  double t49 = t29 + t39;
  double t50 = t31 + t40;
  double t54 = t41 + t45;
  double t55 = t42 + t46;
  double t51 = t21 * t48;
  double t52 = t22 * t47;
  double t56 = r1 * t55;
  double t57 = r3 * t54;
  double t59 = t21 * t55;
  double t60 = t22 * t54;
  double t53 = -t52;
  double t58 = -t57;
  double t61 = -t59;
  double t62 = t60 + t61;
  double t64 = t51 + t53 + t56 + t58;
  double t63 = exp(t62);
  H[0] = (t8 * t26 * t63 * (t64 * t64)) / 2.0 -
         (t8 * t26 * t63 * (r1 * t48 * 2.0 - r3 * t47 * 2.0)) / 2.0;
  H[1] =
      t8 * t26 * t63 * (r2 * t48 + r1 * t50 - r4 * t47 - r3 * t49) *
          (-1.0 / 2.0) +
      (t8 * t26 * t63 * t64 * (r2 * t55 - r4 * t54 + t21 * t50 - t22 * t49)) /
          2.0;
}

__device__ inline void d3f_fun(double* d3f, double* x, double mu1, double mu2,
                               double s1, double s2, double s3, int zono_idx) {
  // this function only returns dddf/dxdxdx and dddf/dxdydx.
  double x1 = x[0];
  double x2 = x[1];

  //     double mu1 = mu_sigma[0];
  //     double mu2 = mu_sigma[1];
  //     double s1 = mu_sigma[2];
  //     double s2 = mu_sigma[3];
  //     double s3 = mu_sigma[4];

  double r1 = cos(dev_rot_angle[zono_idx]);
  double r2 = sin(dev_rot_angle[zono_idx]);
  double r3 = -r2;
  double r4 = r1;

  double t2 = s1 * s3;
  double t3 = r1 * x1;
  double t4 = r2 * x2;
  double t5 = r3 * x1;
  double t6 = r4 * x2;
  double t7 = s2 * s2;
  double t8 = 1.0 / 3.141592653589793;
  double t9 = -mu1;
  double t10 = -mu2;
  double t11 = mu1 / 2.0;
  double t12 = mu2 / 2.0;
  double t13 = -t7;
  double t14 = t3 / 2.0;
  double t15 = t4 / 2.0;
  double t16 = t5 / 2.0;
  double t17 = t6 / 2.0;
  double t18 = -t11;
  double t19 = -t12;
  double t21 = t3 + t4 + t9;
  double t22 = t5 + t6 + t10;
  double t20 = t2 + t13;
  double t35 = t14 + t15 + t18;
  double t36 = t16 + t17 + t19;
  double t23 = 1.0 / t20;
  double t24 = sqrt(t20);
  double t25 = t24;
  double t27 = (r1 * s2 * t23) / 2.0;
  double t28 = (r1 * s3 * t23) / 2.0;
  double t29 = (r2 * s2 * t23) / 2.0;
  double t30 = (r3 * s1 * t23) / 2.0;
  double t31 = (r2 * s3 * t23) / 2.0;
  double t32 = (r3 * s2 * t23) / 2.0;
  double t33 = (r4 * s1 * t23) / 2.0;
  double t34 = (r4 * s2 * t23) / 2.0;
  double t41 = s2 * t23 * t35;
  double t42 = s3 * t23 * t35;
  double t43 = s1 * t23 * t36;
  double t44 = s2 * t23 * t36;
  double t26 = 1.0 / t25;
  double t37 = -t30;
  double t38 = -t32;
  double t39 = -t33;
  double t40 = -t34;
  double t45 = -t43;
  double t46 = -t44;
  double t47 = t27 + t37;
  double t48 = t28 + t38;
  double t49 = t29 + t39;
  double t50 = t31 + t40;
  double t60 = t41 + t45;
  double t61 = t42 + t46;
  double t51 = r1 * t48 * 2.0;
  double t52 = r3 * t47 * 2.0;
  double t54 = t21 * t48;
  double t55 = t21 * t50;
  double t56 = t22 * t47;
  double t57 = t22 * t49;
  double t62 = r1 * t61;
  double t63 = r3 * t60;
  double t64 = r2 * t61;
  double t65 = r4 * t60;
  double t68 = t21 * t61;
  double t69 = t22 * t60;
  double t53 = -t52;
  double t58 = -t56;
  double t59 = -t57;
  double t66 = -t63;
  double t67 = -t65;
  double t70 = -t68;
  double t71 = t51 + t53;
  double t72 = t69 + t70;
  double t74 = t54 + t58 + t62 + t66;
  double t75 = t55 + t59 + t64 + t67;
  double t73 = exp(t72);

  d3f[0] = t8 * t26 * t73 * (t74 * t74 * t74) * (-1.0 / 2.0) +
           t8 * t26 * t71 * t73 * t74 * (3.0 / 2.0);
  d3f[1] = t8 * t26 * t73 * (t74 * t74) * t75 * (-1.0 / 2.0) +
           (t8 * t26 * t71 * t73 * t75) / 2.0 +
           t8 * t26 * t73 * t74 * (r2 * t48 + r1 * t50 - r4 * t47 - r3 * t49);
}

// get the interval bound of the intergral of the probability density function
// over a 2d triangle
__device__ inline void integrator(double* constr, double* xy0, double* H,
                                  double mu1, double mu2, double s1, double s2,
                                  double s3, int zono_idx, int uplo) {
  double f0 = 0;
  f_fun(f0, xy0, mu1, mu2, s1, s2, s3, zono_idx);

  double df0[2] = {0};
  Jacobian_fun(df0, xy0, mu1, mu2, s1, s2, s3, zono_idx);

  double d2f0[2] = {0};
  Hessian_fun(d2f0, xy0, mu1, mu2, s1, s2, s3, zono_idx);

  double d3f0[2] = {0};
  d3f_fun(d3f0, xy0, mu1, mu2, s1, s2, s3, zono_idx);

  double df1 = df0[0];
  double df2 = df0[1];

  double d2f1 = d2f0[0];
  double d2f2 = d2f0[1];

  double d3f1 = d3f0[0];
  double d3f2 = d3f0[1];

  double H1_1 = H[0];
  double H2_1 = H[1];
  double H2_2 = H[2];

  double dx = dev_dx[zono_idx];
  double dy = dev_dy[zono_idx];
  double dxdp = dev_g_p_x[zono_idx] / dev_cg_p[1];

  double t2 = dxdp * dxdp;

  constr[0] = (dx * dy *
               (f0 + (df1 * dx * uplo) / 3.0 + (df2 * dy * uplo) / 3.0 +
                (H1_1 * (dx * dx)) / 1.2E+1 + (H2_2 * (dy * dy)) / 1.2E+1 +
                (H2_1 * dx * dy) / 1.2E+1)) /
              2.0;
  constr[1] = (dx * dy *
               (df1 * dxdp +
                dxdp * ((d2f1 * dx * uplo) / 3.0 + (d2f2 * dy * uplo) / 3.0))) /
              2.0;
  constr[2] = (dx * dy *
               (d2f1 * t2 +
                t2 * ((d3f1 * dx * uplo) / 3.0 + (d3f2 * dy * uplo) / 3.0))) /
              2.0;

  //     constr[0] = df0[1];
  //     constr[1] = df0[2];
}

#endif

//
// Header PROB_FUNCTION_INFO_FRS_FLZONO_H End
//

#include <unistd.h>

#include <cstdio>
#include <cstdlib>

#include "prob_integ.hpp"

// #define MAX_PATCHES_NUM 1024    // This is maximum patch number per zono!
// Otherwise need to change the sum_over kernel
#define BLOCK_SIZE1 32   // block size for integration kernel
#define BLOCK_SIZE2 1024 // block size for sum_over kernel

__global__ void __launch_bounds__(BLOCK_SIZE1, 1)
    eval_patch_kernel(double* x0, double* y0, double* block_inzono_list,
                      double* H1, double* H2, double* H4, double* mu_sigma,
                      double* constr, double* dconstr, double* d2constr) {
  int patch_id =
      (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

  double H[3] = {H1[patch_id], H2[patch_id], H4[patch_id]};

  // throw xy0, mu and sigma into shared memory
  __shared__ volatile double shared_mu_sigma[5];
  __shared__ volatile double shared_xy0[2];

  if (threadIdx.x < 5) {
    shared_mu_sigma[threadIdx.x] = mu_sigma[blockIdx.y * 5 + threadIdx.x];
    if (threadIdx.x == 0) {
      shared_xy0[0] =
          x0[blockIdx.y] + dev_g_p_x[blockIdx.y] * dev_p_slice_beta[0];
      shared_xy0[1] = y0[blockIdx.y];
    }
  }
  __syncthreads();

  // actual computation
  double x[2]; // vertex of the right triangle in a simplex
  int pid_inzono = (int)block_inzono_list[patch_id];
  if (pid_inzono < 0)
    return;
  int xid = pid_inzono % dev_grid_size[0];
  int yid = pid_inzono / dev_grid_size[0];

  // lower right triangle
  double constr1[3];
  x[0] = shared_xy0[0] + xid * dev_dx[blockIdx.y];
  x[1] = shared_xy0[1] + yid * dev_dy[blockIdx.y];
  integrator(constr1, x, H, shared_mu_sigma[0], shared_mu_sigma[1],
             shared_mu_sigma[2], shared_mu_sigma[3], shared_mu_sigma[4],
             blockIdx.y, 1);

  //     double bla[2];
  //     Hessian_fun(bla, x, shared_mu_sigma[0], shared_mu_sigma[1],
  //     shared_mu_sigma[2],
  //                     shared_mu_sigma[3], shared_mu_sigma[4], zono_id);

  // upper right triangle
  double constr2[3];
  x[0] += dev_dx[blockIdx.y];
  x[1] += dev_dy[blockIdx.y];
  integrator(constr2, x, H, shared_mu_sigma[0], shared_mu_sigma[1],
             shared_mu_sigma[2], shared_mu_sigma[3], shared_mu_sigma[4],
             blockIdx.y, -1);

  constr[patch_id] = constr1[0] + constr2[0];
  dconstr[patch_id] = constr1[1] + constr2[1];
  d2constr[patch_id] = constr1[2] + constr2[2];

  //     constr[patch_id] = H[0];
  //     dconstr[patch_id] = H[1];
  //     d2constr[patch_id] = H[2];
}

__global__ void sum_over(double* input, double* output, int len) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * BLOCK_SIZE2 + tid;

  __shared__ volatile double shared_sum[BLOCK_SIZE2];

  if (idx < len)
    shared_sum[tid] = input[idx];
  else
    shared_sum[tid] = 0;
  __syncthreads();

  if (BLOCK_SIZE2 >= 1024) {
    if (tid < 512) {
      shared_sum[tid] += shared_sum[tid + 512];
    }
    __syncthreads();
  }
  if (BLOCK_SIZE2 >= 512) {
    if (tid < 256) {
      shared_sum[tid] += shared_sum[tid + 256];
    }
    __syncthreads();
  }
  if (BLOCK_SIZE2 >= 256) {
    if (tid < 128) {
      shared_sum[tid] += shared_sum[tid + 128];
    }
    __syncthreads();
  }
  if (BLOCK_SIZE2 >= 128) {
    if (tid < 64) {
      shared_sum[tid] += shared_sum[tid + 64];
    }
    __syncthreads();
  }
  if (tid < 32) {
    if (BLOCK_SIZE2 >= 64)
      shared_sum[tid] += shared_sum[tid + 32];
    if (BLOCK_SIZE2 >= 32)
      shared_sum[tid] += shared_sum[tid + 16];
    if (BLOCK_SIZE2 >= 16)
      shared_sum[tid] += shared_sum[tid + 8];
    if (BLOCK_SIZE2 >= 8)
      shared_sum[tid] += shared_sum[tid + 4];
    if (BLOCK_SIZE2 >= 4)
      shared_sum[tid] += shared_sum[tid + 2];
    if (BLOCK_SIZE2 >= 2)
      shared_sum[tid] += shared_sum[tid + 1];
  }

  if (tid == 0) {
    output[bid] = shared_sum[0];
  }
}

// helper functions
inline void initializeDeviceArray(double** dev_arr, int size) {
  hipMalloc((void**)dev_arr, size * sizeof(double));
  hipMemset(*dev_arr, 0, size * sizeof(double));
}

// inline void initializeDeviceArray(double** dev_arr, double* host_arr, int
// size) {
//     hipMalloc((void**)dev_arr, size * sizeof(double));
//     hipMemcpy(*dev_arr, host_arr, size * sizeof(double),
//     hipMemcpyHostToDevice);
// }
inline void initializeDeviceArray(double** dev_arr, const double* host_arr,
                                  int size) {
  hipMalloc((void**)dev_arr, size * sizeof(double));
  hipMemcpy(*dev_arr, host_arr, size * sizeof(double), hipMemcpyHostToDevice);
}

inline int divideup(int a, int b) {
  int c = a / b;
  if (c * b == a)
    return c;
  return c + 1;
}

/// Performs the integration of the probability function over the zonotopes.
///
/// @param[in] inputs The inputs to the integration, see the member variable
/// documentation for more information.
///
/// @param[out] outputs The outputs of the integration, see the member variable
/// documentation for more information.
namespace roahm::prob_integ {
::roahm::ProbIntegrationOutputs
ProbIntegration(const ::roahm::ProbIntegrationInputs& inputs) {
  const int num_zono = inputs.num_zonos_;
  const double* host_x0 = inputs.x0_;
  const double* host_y0 = inputs.y0_;
  const double* host_dx = inputs.dx_;
  const double* host_dy = inputs.dy_;
  const double* host_block_inzono_list = inputs.block_inzono_list_;
  const double* host_H1 = inputs.H1_;
  const double* host_H2 = inputs.H2_;
  const double* host_H4 = inputs.H4_;
  const double* host_rot_angle = inputs.rot_angle_;
  const double* host_mu_sigma = inputs.mu_sigma_;
  const double* cg_p = inputs.cg_p_;
  const double* host_g_p_x = inputs.g_p_x_;
  const int grid_size = inputs.grid_size_;
  const double p = inputs.p_;
  const int total_num_patch = num_zono * grid_size * grid_size;

  hipMemcpyToSymbol(HIP_SYMBOL(dev_grid_size), &grid_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_zono), &num_zono, sizeof(int));

  double* dev_x0 = nullptr;
  initializeDeviceArray(&dev_x0, host_x0, num_zono);

  double* dev_y0 = nullptr;
  initializeDeviceArray(&dev_y0, host_y0, num_zono);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), host_dx, num_zono * sizeof(double));

  hipMemcpyToSymbol(HIP_SYMBOL(dev_dy), host_dy, num_zono * sizeof(double));

  double* dev_block_inzono_list = nullptr;
  initializeDeviceArray(&dev_block_inzono_list, host_block_inzono_list,
                        num_zono * grid_size * grid_size);

  double* dev_H1 = nullptr;
  initializeDeviceArray(&dev_H1, host_H1, total_num_patch);

  double* dev_H2 = nullptr;
  initializeDeviceArray(&dev_H2, host_H2, total_num_patch);

  double* dev_H4 = nullptr;
  initializeDeviceArray(&dev_H4, host_H4, total_num_patch);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_rot_angle), host_rot_angle, num_zono * sizeof(double));

  double* dev_mu_sigma = nullptr;
  initializeDeviceArray(&dev_mu_sigma, host_mu_sigma, 5 * num_zono);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_cg_p), cg_p, 2 * sizeof(double));

  hipMemcpyToSymbol(HIP_SYMBOL(dev_g_p_x), host_g_p_x, num_zono * sizeof(double));

  hipMemcpyToSymbol(HIP_SYMBOL(dev_p_slice_beta), &p, sizeof(double));

  // Setup output structure
  ::roahm::ProbIntegrationOutputs outputs{};

  // Point variables to the output structur member variables, not cleanest
  // but prevents needing to change anything below.
  double* constr = &outputs.constraint_val_;
  double* dconstr = &outputs.d_constraint_val_;
  double* d2constr = &outputs.d2_constraint_val_;
  float* gpu_time = &outputs.computation_time_;

  hipEvent_t start, stop;

  // define output in device
  double* dev_constr_1 = nullptr;
  initializeDeviceArray(&dev_constr_1, total_num_patch);
  double* dev_dconstr_1 = nullptr;
  initializeDeviceArray(&dev_dconstr_1, total_num_patch);
  double* dev_d2constr_1 = nullptr;
  initializeDeviceArray(&dev_d2constr_1, total_num_patch);

  int sum_length = total_num_patch;
  int num_sum_block = divideup(sum_length, BLOCK_SIZE2);
  double* dev_constr_2 = nullptr;
  initializeDeviceArray(&dev_constr_2, num_sum_block);
  double* dev_dconstr_2 = nullptr;
  initializeDeviceArray(&dev_dconstr_2, num_sum_block);
  double* dev_d2constr_2 = nullptr;
  initializeDeviceArray(&dev_d2constr_2, num_sum_block);

  //////  MAIN STARTS HERE //////
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 dimGrid(divideup(grid_size * grid_size, BLOCK_SIZE1), num_zono);
  eval_patch_kernel<<<dimGrid, BLOCK_SIZE1>>>(
      dev_x0, dev_y0, dev_block_inzono_list, dev_H1, dev_H2, dev_H4,
      dev_mu_sigma, dev_constr_1, dev_dconstr_1, dev_d2constr_1);
  hipDeviceSynchronize();
  hipDeviceSynchronize();

  // sum up all integrations
  bool final_summation_in_1 = true;
  while (sum_length > 1) {
    sum_over<<<num_sum_block, BLOCK_SIZE2>>>(dev_constr_1, dev_constr_2,
                                             sum_length);
    sum_over<<<num_sum_block, BLOCK_SIZE2>>>(dev_dconstr_1, dev_dconstr_2,
                                             sum_length);
    sum_over<<<num_sum_block, BLOCK_SIZE2>>>(dev_d2constr_1, dev_d2constr_2,
                                             sum_length);
    final_summation_in_1 = false;

    sum_length = num_sum_block;
    num_sum_block = divideup(sum_length, BLOCK_SIZE2);
    if (sum_length > 1) {
      sum_over<<<num_sum_block, BLOCK_SIZE2>>>(dev_constr_2, dev_constr_1,
                                               sum_length);
      sum_over<<<num_sum_block, BLOCK_SIZE2>>>(dev_dconstr_2, dev_dconstr_1,
                                               sum_length);
      sum_over<<<num_sum_block, BLOCK_SIZE2>>>(dev_d2constr_2, dev_d2constr_1,
                                               sum_length);
      sum_length = num_sum_block;
      final_summation_in_1 = true;
    }
  }

  // copy output
  if (final_summation_in_1) {
    hipMemcpy(constr, dev_constr_1, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dconstr, dev_dconstr_1, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(d2constr, dev_d2constr_1, sizeof(double),
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(constr, dev_constr_2, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dconstr, dev_dconstr_2, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(d2constr, dev_d2constr_2, sizeof(double),
               hipMemcpyDeviceToHost);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(gpu_time, start, stop);
  // printf("Time spent: %.5f ms\n", *gpu_time);
  //////  MAIN ENDS HERE //////

  // free memory
  hipFree(dev_x0);
  hipFree(dev_y0);
  hipFree(dev_block_inzono_list);
  hipFree(dev_mu_sigma);
  hipFree(dev_H1);
  hipFree(dev_H2);
  hipFree(dev_H4);
  hipFree(dev_constr_1);
  hipFree(dev_dconstr_1);
  hipFree(dev_d2constr_1);
  hipFree(dev_constr_2);
  hipFree(dev_dconstr_2);
  hipFree(dev_d2constr_2);
  return outputs;
}
} // namespace roahm::prob_integ

#undef BLOCK_SIZE1
#undef BLOCK_SIZE2