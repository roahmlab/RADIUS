#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "cuda_interval_lib.hpp"
#include "pre_slice.hpp"

using CudaInterval = interval_gpu<double>;
#define MAX_ZONO_NUM 1000

#define BLOCK_SIZE1 512 // block size for slicing kernel
#define BLOCK_SIZE2 32  // block size for hessian kernel

__constant__ double dev_u0v0r0_slice_beta[3];
__constant__ int dev_num_zono[1];
__constant__ int dev_grid_size[1];
__constant__ double dev_dx[MAX_ZONO_NUM];
__constant__ double dev_dy[MAX_ZONO_NUM];
__constant__ double dev_rot_angle[MAX_ZONO_NUM];
__constant__ double dev_g_p_x[MAX_ZONO_NUM];

// __device__ inline void Hessian_fun(CudaInterval* H, Interval* x, double*
// mu_sigma, int zono_id) {
__device__ inline void Hessian_fun(CudaInterval* H, CudaInterval* x, double mu1,
                                   double mu2, double s1_1, double s2_1,
                                   double s2_2, int zono_id) {
  CudaInterval x1 = x[0];
  CudaInterval x2 = x[1];

  //     double mu1 = mu_sigma[0];
  //     double mu2 = mu_sigma[1];
  //     double s1_1 = mu_sigma[2];
  //     double s2_1 = mu_sigma[3];
  //     double s2_2 = mu_sigma[4];

  double R1_1 = cos(dev_rot_angle[zono_id]);
  double R1_2 = -sin(dev_rot_angle[zono_id]);
  double R2_1 = -R1_2;
  double R2_2 = R1_1;

  CudaInterval t2 = R1_1 * x1;
  CudaInterval t3 = R1_2 * x1;
  CudaInterval t4 = R2_1 * x2;
  CudaInterval t5 = R2_2 * x2;
  double t6 = s1_1 * s2_2;
  double t7 = s2_1 * s2_1;
  double t8 = 1.0 / 3.141592653589793;
  double t9 = -mu1;
  double t10 = -mu2;
  double t11 = mu1 / 2.0;
  double t12 = mu2 / 2.0;
  CudaInterval t13 = t2 / 2.0;
  CudaInterval t14 = t3 / 2.0;
  CudaInterval t15 = t4 / 2.0;
  CudaInterval t16 = t5 / 2.0;
  CudaInterval t17 = -t7;
  CudaInterval t18 = -t11;
  CudaInterval t19 = -t12;
  CudaInterval t21 = t2 + t4 + t9;
  CudaInterval t22 = t3 + t5 + t10;
  CudaInterval t20 = t6 + t17;
  CudaInterval t35 = t13 + t15 + t18;
  CudaInterval t36 = t14 + t16 + t19;
  CudaInterval t23 = 1.0 / t20;
  CudaInterval t24 = sqrt(t20);
  CudaInterval t25 = t24;
  CudaInterval t27 = (R1_2 * s1_1 * t23) / 2.0;
  CudaInterval t28 = (R1_1 * s2_1 * t23) / 2.0;
  CudaInterval t29 = (R1_1 * s2_2 * t23) / 2.0;
  CudaInterval t30 = (R1_2 * s2_1 * t23) / 2.0;
  CudaInterval t31 = (R2_2 * s1_1 * t23) / 2.0;
  CudaInterval t32 = (R2_1 * s2_1 * t23) / 2.0;
  CudaInterval t33 = (R2_1 * s2_2 * t23) / 2.0;
  CudaInterval t34 = (R2_2 * s2_1 * t23) / 2.0;
  CudaInterval t41 = s1_1 * t23 * t36;
  CudaInterval t42 = s2_1 * t23 * t35;
  CudaInterval t43 = s2_2 * t23 * t35;
  CudaInterval t44 = s2_1 * t23 * t36;
  CudaInterval t26 = 1.0 / t25;
  CudaInterval t37 = -t28;
  CudaInterval t38 = -t30;
  CudaInterval t39 = -t32;
  CudaInterval t40 = -t34;
  CudaInterval t45 = -t42;
  CudaInterval t46 = -t44;
  CudaInterval t47 = t27 + t37;
  CudaInterval t48 = t29 + t38;
  CudaInterval t49 = t31 + t39;
  CudaInterval t50 = t33 + t40;
  CudaInterval t59 = t41 + t45;
  CudaInterval t60 = t43 + t46;
  CudaInterval t51 = R2_2 * t47;
  CudaInterval t52 = R1_2 * t49;
  CudaInterval t53 = R2_1 * t48;
  CudaInterval t54 = R1_1 * t50;
  CudaInterval t55 = t22 * t47;
  CudaInterval t56 = t21 * t48;
  CudaInterval t57 = t22 * t49;
  CudaInterval t58 = t21 * t50;
  CudaInterval t61 = R1_2 * t59;
  CudaInterval t62 = R1_1 * t60;
  CudaInterval t63 = R2_2 * t59;
  CudaInterval t64 = R2_1 * t60;
  CudaInterval t65 = t22 * t59;
  CudaInterval t66 = t21 * t60;
  CudaInterval t67 = -t65;
  CudaInterval t68 = -t66;
  CudaInterval t69 = t51 + t52 + t53 + t54;
  CudaInterval t72 = t55 + t56 + t61 + t62;
  CudaInterval t73 = t57 + t58 + t63 + t64;
  CudaInterval t70 = t67 + t68;
  CudaInterval t71 = exp(t70);
  CudaInterval t74 = (t8 * t26 * t69 * t71) / 2.0;
  CudaInterval t76 = (t8 * t26 * t71 * t72 * t73) / 2.0;
  CudaInterval t75 = -t74;
  CudaInterval t77 = t75 + t76;
  H[0] = t8 * t26 * t71 * (R1_1 * t48 * 2.0 + R1_2 * t47 * 2.0) * (-1.0 / 2.0) +
         (t8 * t26 * t71 * (t72 * t72)) / 2.0;
  H[1] = t77;
  H[2] = t8 * t26 * t71 * (R2_1 * t50 * 2.0 + R2_2 * t49 * 2.0) * (-1.0 / 2.0) +
         (t8 * t26 * t71 * (t73 * t73)) / 2.0;
}

__global__ void __launch_bounds__(BLOCK_SIZE2, 1)
    hessian_kernel(double* x0, double* y0, double* mu_sigma, double* H1,
                   double* H2, double* H4, double* block_inzono_list) {
  int zono_id = blockIdx.y;
  int patch_id = (zono_id * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

  // get mu, sigma, and xy0 for zonotope #zono_id
  __shared__ volatile double shared_mu_sigma[5];
  __shared__ volatile double shared_xy0[2];
  if (threadIdx.x < 5) {
    shared_mu_sigma[threadIdx.x] = mu_sigma[zono_id * 5 + threadIdx.x];
    if (threadIdx.x == 0) {
      shared_xy0[0] = x0[zono_id];
      shared_xy0[1] = y0[zono_id];
    }
  }
  __syncthreads();

  // actual computation
  CudaInterval x[2];
  CudaInterval H[3];
  int pid_inzono = (int)block_inzono_list[patch_id];
  if (pid_inzono < 0)
    return;
  int xid = pid_inzono % dev_grid_size[0];
  int yid = pid_inzono / dev_grid_size[0];
  x[1] = CudaInterval(shared_xy0[1] + yid * dev_dy[zono_id],
                      shared_xy0[1] + (yid + 1) * dev_dy[zono_id]);

  double xlo = shared_xy0[0] + xid * dev_dx[zono_id] - dev_g_p_x[zono_id];
  double dxx = (2 * dev_g_p_x[zono_id] + dev_dx[zono_id]) / 12;
  double H_ub[3] = {nanf(""), nanf(""), nanf("")};
  for (int i = 0; i < 12; i++) {
    x[0] = CudaInterval(xlo, xlo + dxx);
    xlo += dxx;
    Hessian_fun(H, x, shared_mu_sigma[0], shared_mu_sigma[1],
                shared_mu_sigma[2], shared_mu_sigma[3], shared_mu_sigma[4],
                zono_id);
    H_ub[0] = fmax(H_ub[0], H[0].upper());
    H_ub[1] = fmax(H_ub[1], H[1].upper());
    H_ub[2] = fmax(H_ub[2], H[2].upper());
  }

  H1[patch_id] = H_ub[0];
  H2[patch_id] = H_ub[1];
  H4[patch_id] = H_ub[2];
}

__global__ void __launch_bounds__(BLOCK_SIZE1, 1)
    slice_u0v0r0_kernel(double* x0, double* y0, double* g_u0_x, double* g_u0_y,
                        double* g_v0_x, double* g_v0_y, double* g_r0_x,
                        double* g_r0_y) {
  int zono_id = threadIdx.x + blockDim.x * blockIdx.x;
  if (zono_id >= dev_num_zono[0])
    return;

  x0[zono_id] = x0[zono_id] + dev_u0v0r0_slice_beta[0] * g_u0_x[zono_id] +
                dev_u0v0r0_slice_beta[1] * g_v0_x[zono_id] +
                dev_u0v0r0_slice_beta[2] * g_r0_x[zono_id];
  y0[zono_id] = y0[zono_id] + dev_u0v0r0_slice_beta[0] * g_u0_y[zono_id] +
                dev_u0v0r0_slice_beta[1] * g_v0_y[zono_id] +
                dev_u0v0r0_slice_beta[2] * g_r0_y[zono_id];
}

inline void initializeDeviceArray(double** dev_arr, int size) {
  hipMalloc((void**)dev_arr, size * sizeof(double));
  hipMemset(*dev_arr, 0, size * sizeof(double));
}

inline void initializeDeviceArray(double** dev_arr, const double* host_arr,
                                  int size) {
  hipMalloc((void**)dev_arr, size * sizeof(double));
  hipMemcpy(*dev_arr, host_arr, size * sizeof(double), hipMemcpyHostToDevice);
}

inline int divideup(int a, int b) {
  int c = a / b;
  if (c * b == a)
    return c;
  return c + 1;
}

namespace roahm::pre_slice {
PreSliceOutputs
PreSliceImpl(const int grid_size, const int num_zono, const double* host_x0,
             const double* host_y0, const double* host_dx,
             const double* host_dy, const double* host_u0v0r0_slice_beta,
             const double* host_g_u0_x, const double* host_g_u0_y,
             const double* host_g_v0_x, const double* host_g_v0_y,
             const double* host_g_r0_x, const double* host_g_r0_y,
             const double* host_block_inzono_list, const double* host_rot_angle,
             const double* host_mu_sigma, const double* host_g_p_x) {
  hipMemcpyToSymbol(HIP_SYMBOL(dev_grid_size), &grid_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_zono), &num_zono, sizeof(int));

  double* dev_x0 = nullptr;
  initializeDeviceArray(&dev_x0, host_x0, num_zono);

  double* dev_y0 = nullptr;
  initializeDeviceArray(&dev_y0, host_y0, num_zono);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), host_dx, num_zono * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_dy), host_dy, num_zono * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_u0v0r0_slice_beta), host_u0v0r0_slice_beta,
                     3 * sizeof(double));

  double* dev_g_u0_x = nullptr;
  initializeDeviceArray(&dev_g_u0_x, host_g_u0_x, num_zono);

  double* dev_g_u0_y = nullptr;
  initializeDeviceArray(&dev_g_u0_y, host_g_u0_y, num_zono);

  double* dev_g_v0_x = nullptr;
  initializeDeviceArray(&dev_g_v0_x, host_g_v0_x, num_zono);

  double* dev_g_v0_y = nullptr;
  initializeDeviceArray(&dev_g_v0_y, host_g_v0_y, num_zono);

  double* dev_g_r0_x = nullptr;
  initializeDeviceArray(&dev_g_r0_x, host_g_r0_x, num_zono);

  double* dev_g_r0_y = nullptr;
  initializeDeviceArray(&dev_g_r0_y, host_g_r0_y, num_zono);

  double* dev_block_inzono_list = nullptr;
  initializeDeviceArray(&dev_block_inzono_list, host_block_inzono_list,
                        num_zono * grid_size * grid_size);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_rot_angle), host_rot_angle, num_zono * sizeof(double));

  double* dev_mu_sigma = nullptr;
  initializeDeviceArray(&dev_mu_sigma, host_mu_sigma, 5 * num_zono);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_g_p_x), host_g_p_x, num_zono * sizeof(double));

  // TODO
  // set output [x0, y0, H1, H2, H4, t]
  PreSliceOutputs outputs{
      std::shared_ptr<double>{new double[num_zono]},
      std::shared_ptr<double>{new double[num_zono]},
      std::shared_ptr<double>{new double[num_zono * grid_size * grid_size]},
      std::shared_ptr<double>{new double[num_zono * grid_size * grid_size]},
      std::shared_ptr<double>{new double[num_zono * grid_size * grid_size]},
      std::shared_ptr<float>{new float[1]},
  };
  double* x0 = outputs.x0_.get();
  double* y0 = outputs.y0_.get();
  double* H1 = outputs.H1_.get();
  double* H2 = outputs.H2_.get();
  double* H4 = outputs.H4_.get();
  float* gpu_time = outputs.gpu_time_.get();

  hipEvent_t start, stop;
  // define output in device
  double* dev_H1 = nullptr;
  initializeDeviceArray(&dev_H1, num_zono * grid_size * grid_size);
  double* dev_H2 = nullptr;
  initializeDeviceArray(&dev_H2, num_zono * grid_size * grid_size);
  double* dev_H4 = nullptr;
  initializeDeviceArray(&dev_H4, num_zono * grid_size * grid_size);

  //////  MAIN STARTS HERE //////
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  slice_u0v0r0_kernel<<<divideup(num_zono, BLOCK_SIZE1), BLOCK_SIZE1>>>(
      dev_x0, dev_y0, dev_g_u0_x, dev_g_u0_y, dev_g_v0_x, dev_g_v0_y,
      dev_g_r0_x, dev_g_r0_y);

  dim3 dimGrid(divideup(grid_size * grid_size, BLOCK_SIZE2), num_zono);
  hessian_kernel<<<dimGrid, BLOCK_SIZE2>>>(dev_x0, dev_y0, dev_mu_sigma, dev_H1,
                                           dev_H2, dev_H4,
                                           dev_block_inzono_list);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(gpu_time, start, stop);
  // printf("Time spent: %.5f ms\n", *gpu_time);
  //////  MAIN ENDS HERE //////

  // copy output
  hipMemcpy(x0, dev_x0, num_zono * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(y0, dev_y0, num_zono * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(H1, dev_H1, num_zono * grid_size * grid_size * sizeof(double),
             hipMemcpyDeviceToHost);
  hipMemcpy(H2, dev_H2, num_zono * grid_size * grid_size * sizeof(double),
             hipMemcpyDeviceToHost);
  hipMemcpy(H4, dev_H4, num_zono * grid_size * grid_size * sizeof(double),
             hipMemcpyDeviceToHost);

  // free memory
  hipFree(dev_x0);
  hipFree(dev_y0);
  hipFree(dev_g_u0_x);
  hipFree(dev_g_u0_y);
  hipFree(dev_g_v0_x);
  hipFree(dev_g_v0_y);
  hipFree(dev_g_r0_x);
  hipFree(dev_g_r0_y);
  hipFree(dev_block_inzono_list);
  hipFree(dev_mu_sigma);
  hipFree(dev_H1);
  hipFree(dev_H2);
  hipFree(dev_H4);
  return outputs;
}

PreSliceOutputs PreSlice(const ::roahm::CudaInfo& cuda_info,
                         const std::array<double, 3>& u0v0r0_slice_beta_in,
                         const double* const mu_sigma) {
  return PreSliceImpl(
      cuda_info.grid_size_, cuda_info.num_zono_, cuda_info.grid_x0_.data(),
      cuda_info.grid_y0_.data(), cuda_info.grid_dx_.data(),
      cuda_info.grid_dy_.data(), u0v0r0_slice_beta_in.data(),
      cuda_info.g_u0_x_.data(), cuda_info.g_u0_y_.data(),
      cuda_info.g_v0_x_.data(), cuda_info.g_v0_y_.data(),
      cuda_info.g_r0_x_.data(), cuda_info.g_r0_y_.data(),
      cuda_info.block_inzono_list_.data(), cuda_info.rot_angle_.data(),
      mu_sigma, cuda_info.g_p_x_.data());
}

} // namespace roahm::pre_slice

#undef MAX_ZONO_NUM
#undef BLOCK_SIZE1
#undef BLOCK_SIZE2
